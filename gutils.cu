#include "hip/hip_runtime.h"
#include "gutils.hh"
#include <TStopwatch.h>
#include <iostream>
#include <iomanip>
#include <functional>
#include <numeric>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <Gpufit/gpufit.h>

using namespace thrust;

struct exp_prod_functor : public binary_function<float, float, float>{
  __host__ __device__ float operator()(const float& x, const float& y) const{
    return expf(x*y);
  }
};

struct square_sum_functor : public binary_function<float, float, float>{
  __host__ __device__ float operator()(const float& x, const float& y) const{
    return x*x+y*y;
  }
};

struct prod_log_functor : public binary_function<float, float, float>{
  __host__ __device__ float operator()(const float& x, const float& y) const{
    return x*logf(y);
  }
};

struct log_sum_functor : public binary_function<float, float, float>{
  __host__ __device__ float operator()(const float& x, const float& y) const{
    return logf(x)+logf(y);
  }
};

struct saxpy_functor{
  const float a;
  saxpy_functor(float _a) : a(_a) {}
  __host__ __device__ float operator()(const float& x, const float& y){
    return x+a*y;
  }
};

struct sqrt_norm_functor{
  const float a;
  sqrt_norm_functor(float _a) : a(_a) {}
  __host__ __device__ float operator()(const float& x){
    return sqrtf(x/a);
  }
};

// maxima of waveforms in x with keys k
std::vector<float> Maxima(const device_vector<float>& x,
			  const device_vector<int>& k,
			  device_vector<float>& y, device_vector<int>& s){
  pair<device_vector<int>::iterator, device_vector<float>::iterator> p =
    reduce_by_key(k.begin(), k.end(), x.begin(), s.begin(), y.begin(),
		  equal_to<int>(), maximum<float>());
  std::vector<float> v(p.first-s.begin());
  thrust::copy(y.begin(), y.begin()+v.size(), v.begin());
  return v;
}

// maxima in waveforms in x with keys k along with the index of the maxima
// pass the number of waveforms n to avoid having to count keys
std::pair<std::vector<float>,
	  std::vector<int> > MaxWithIndex(const device_vector<float>& x,
					  const device_vector<int>& k,
					  const device_vector<int>& t,
					  device_vector<float>& y,
					  device_vector<int>& s, int n){
  zip_iterator<tuple<device_vector<const float>::iterator,
		     device_vector<const int>::iterator> >
    z0 = make_zip_iterator(make_tuple(x.begin(), t.begin()));
  zip_iterator<tuple<device_vector<float>::iterator,
		     device_vector<int>::iterator> > 
    z1 = make_zip_iterator(make_tuple(y.begin(), s.begin()));
  reduce_by_key(k.begin(), k.end(), z0, make_discard_iterator(), z1,
		equal_to<int>(), maximum<tuple<float, int> >());
  // fixme: the lines below aren't actually accessing the reduced values
  std::vector<float> v(n);
  thrust::copy(y.begin(), y.begin()+n, v.begin());
  std::vector<int>   w(n);
  thrust::copy(s.begin(), s.begin()+n, w.begin());
  return std::make_pair(v, w);
}

// minima of waveforms in x with keys k
std::vector<float> Minima(const device_vector<float>& x,
			  const device_vector<int>& k,
			  device_vector<float>& y, device_vector<int>& s){
  pair<device_vector<int>::iterator, device_vector<float>::iterator> p =
    reduce_by_key(k.begin(), k.end(), x.begin(), s.begin(), y.begin(),
		  equal_to<int>(), minimum<float>());
  std::vector<float> v(p.first-s.begin());
  thrust::copy(y.begin(), y.begin()+v.size(), v.begin());
  return v;
}

// minima in waveforms in x with keys k along with the index of the minima
// pass the number of waveforms n to avoid having to count keys
std::pair<std::vector<float>,
	  std::vector<int> > MinWithIndex(const device_vector<float>& x,
					  const device_vector<int>& k,
					  const device_vector<int>& t,
					  device_vector<float>& y,
					  device_vector<int>& s, int n){
  zip_iterator<tuple<device_vector<const float>::iterator,
		     device_vector<const int>::iterator> >
    z0 = make_zip_iterator(make_tuple(x.begin(), t.begin()));
  zip_iterator<tuple<device_vector<float>::iterator,
		     device_vector<int>::iterator> >
    z1 = make_zip_iterator(make_tuple(y.begin(), s.begin()));
  reduce_by_key(k.begin(), k.end(), z0, make_discard_iterator(), z1,
		equal_to<int>(), minimum<tuple<float, int> >());
  std::vector<float> v(n);
  thrust::copy(y.begin(), y.begin()+n, v.begin());
  std::vector<int>   w(n);
  thrust::copy(s.begin(), s.begin()+n, w.begin());
  return std::make_pair(v, w);
}
  
// integral of values in x for keys k from n to n+m in each wf starting at s
std::vector<float> MeanVals(const device_vector<float>& x,
			    const device_vector<int>& k,
			    device_vector<float>& y, int n, int m,
			    std::vector<int>& s, bool do_sum=true){
  if(do_sum) inclusive_scan_by_key(k.begin(), k.end(), x.begin(), y.begin());
  std::vector<float> v(s.size());
  std::vector<float> tmp(y.size());
  thrust::copy(y.begin(), y.end(), tmp.begin());
  for(unsigned i=0; i<v.size(); i++)
    v[i] = (tmp[s[i]+m+n] - tmp[s[i]+m])/n;
  return v;
}

// integral of values in x for keys k from n to n+m in each wf starting at s
std::vector<float> MeanVals(const device_vector<float>& x,
			    const device_vector<int>& k,
			    device_vector<float>& y,
			    std::vector<int>& n, std::vector<int>& m,
			    std::vector<int>& s, bool do_sum=true){
  if(do_sum) inclusive_scan_by_key(k.begin(), k.end(), x.begin(), y.begin());
  std::vector<float> v(s.size());
  std::vector<float> tmp(y.size());
  thrust::copy(y.begin(), y.end(), tmp.begin());
  for(unsigned i=0; i<v.size(); i++)
    v[i] = (tmp[s[i]+m[i]+n[i]] - tmp[s[i]+m[i]]) / n[i];
  return v;
}

// rms of the values in x for keys k from m to m+n in each wf starting at s
std::vector<float> RMSVals(const device_vector<float>& x,
			   const device_vector<int>& k,
			   device_vector<float>& y, int m, int n,
			   std::vector<int>& s, bool do_sum=true){
  if(do_sum) inclusive_scan_by_key(k.begin(), k.end(), x.begin(), y.begin(),
				   equal_to<int>(), square_sum_functor());
  std::vector<float> v(s.size());
  std::vector<float> tmp(y.size());
  thrust::copy(y.begin(), y.end(), tmp.begin());
  for(unsigned i=0; i<v.size(); i++)
    v[i] = sqrt((tmp[s[i]+m+n] - tmp[s[i]+m])/n);
  return v;
}

std::vector<float> ExpDecay(const device_vector<float>& x,
			    const device_vector<float>& y,
			    const device_vector<int>& k,
			    device_vector<float>& z, std::vector<int>& s){
  const int N = (int) x.size();
  const int n = (int) x.size() / s.size();
  std::vector<float> mean(s.size()), tau(s.size()), tmp(s.size());
  inclusive_scan_by_key(k.begin(), k.begin()+N, x.begin(), z.begin(),
			equal_to<int>(), plus<float>());
  thrust::copy(z.begin(), z.begin()+mean.size(), mean.begin());
  std::for_each(mean.begin(), mean.end(), [&](float& v){ v/=n; });
  transform(x.begin(), x.end(), y.begin(), z.begin(), prod_log_functor());
  inclusive_scan_by_key(k.begin(), k.begin()+N, z.begin(), z.begin(),
			equal_to<int>(), plus<float>());
  thrust::copy(z.begin(), z.begin()+tau.size(), tau.begin());
  inclusive_scan_by_key(k.begin(), k.begin()+N, y.begin(), z.begin(),
			equal_to<int>(), log_sum_functor());
  thrust::copy(z.begin(), z.begin()+tmp.size(), tmp.begin());
 for(unsigned i=0; i<tau.size(); i++) tau[i] = tau[i] -  mean[i]*tmp[i];
  inclusive_scan_by_key(k.begin(), k.begin()+N, x.begin(), z.begin(),
			equal_to<int>(), square_sum_functor());
  for(unsigned i=0; i<tau.size(); i++)
    tau[i] = tau[i] / (n*mean[i]*mean[i]-tmp[i]);
  return tau;
}
  
// pole zero correction on x with keys k, pz=1-exp(-sampling/tau)
void PoleZero(device_vector<float>& x,
	      const device_vector<int>& k,
	      device_vector<float>& y, float pz, bool in_place){
  inclusive_scan_by_key(k.begin(), k.end(), x.begin(), y.begin());
  transform(y.begin(), y.end(),
	    make_constant_iterator(pz), y.begin(), multiplies<float>());
  if(in_place)
    transform(y.begin(), y.end(), x.begin(), x.begin(), plus<float>());
  else
    transform(y.begin(), y.end(), x.begin(), y.begin(), plus<float>());
}

// trapezoidal filter from x->y with keys k
// l samples fall time, m samples flat top, n samples rise time
// assumed to be symmetric if last argument is not specified
void Trap(const device_vector<float>& x,
	  const device_vector<int>& k,
	  device_vector<float>& y, device_vector<int>& s, int l,int m,int n=0){
  if(l <= 0 || m < 0) return;
  if(n <= 0) n = l;
  copy(x.begin(), x.end(), y.begin());
  if(n != l) transform(y.begin(), y.end(),make_constant_iterator<float>(1.f/n),
		       y.begin(), multiplies<float>());
  std::vector<int> N({n, m+n, l+m+n});
  std::vector<float> f({-1.f/n, -1.f/l, 1.f/l});
  for(unsigned i=0; i<N.size(); i++){
    transform(k.begin(), k.end()-N[i], k.begin()+N[i],
	      s.begin(), equal_to<int>());
    if(n == l){
      if(i < 2) transform_if(y.begin()+N[i], y.end(), x.begin(), s.begin(),
			     y.begin()+N[i], minus<float>(),identity<float>());
      else transform_if(y.begin()+N[i], y.end(), x.begin(), s.begin(),
			y.begin()+N[i], plus<float>(), identity<float>());
    }
    else transform_if(y.begin()+N[i], y.end(), x.begin(), s.begin(),
		      y.begin()+N[i], saxpy_functor(f[i]), identity<float>());
  }
  if(n == l) transform(y.begin(), y.end(),make_constant_iterator<float>(1.f/n),
		       y.begin(), multiplies<float>());
  inclusive_scan_by_key(k.begin(), k.end(), y.begin(), y.begin());
}

std::map<std::string, MultiWaveform*>
ProcessMultiWaveformGPU(MultiWaveform* wf,
			bool copy_wf){
  TStopwatch ptime;
  ptime.Start();
  std::vector<std::pair<int, int> > blocks = wf->GetBlocks(50000*2048);
  std::map<std::string, MultiWaveform*> mwf;
  std::vector<int> wfstart = wf->GetWFStart();
  if(blocks.size() == 0) return mwf;
  float sampling = wf->GetParam("sampling");
  // transformed waveforms
  MultiWaveform *wfb=NULL, *wfp=NULL, *wfc=NULL;
  MultiWaveform *wfs=NULL, *wff=NULL, *wfa=NULL;
  if(copy_wf){
    wfb = new MultiWaveform(*wf, copy_wf);
    wfp = new MultiWaveform(*wf, copy_wf);
    //wfc = new MultiWaveform(*wf, copy_wf);
    //wfs = new MultiWaveform(*wf, copy_wf);
    //wff = new MultiWaveform(*wf, copy_wf);
    wfa = new MultiWaveform(*wf, copy_wf);
  }
  wfc = new MultiWaveform(*wf, copy_wf);
  wfs = new MultiWaveform(*wf, copy_wf);
  wff = new MultiWaveform(*wf, copy_wf);
  // device vectors
  device_vector<float> y;
  device_vector<int>   k;
  device_vector<int>   t;
  device_vector<int>   s;
  device_vector<float> z;
  device_vector<float> v;
  // process blocks of waveforms until the multiwaveform is fully processed
  for(auto const& pr : blocks){
    int start = wf->GetWFStart(pr.first);
    int end   = wf->GetWFEnd(pr.second);
    std::vector<int> wfst(wfstart.begin()+pr.first, wfstart.begin()+pr.second);
    std::for_each(wfst.begin(), wfst.end(),
		  [&](int& s){ s -= *(wfstart.begin()+pr.first); });
    if(end-start != (int) y.size()){
      y.resize(end-start);
      k.resize(y.size());
      t.resize(y.size());
      s.resize(y.size());
      z.resize(y.size());
      v.resize(y.size());
    }
    copy(wf->wfy.begin()+start, wf->wfy.begin()+end, y.begin());
    copy(wf->wfi.begin()+start, wf->wfi.begin()+end, k.begin());
    std::vector<int> mwfindex(end-start);
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      std::vector<int> tmp(wf->GetWFLength(iwf));
      std::iota(tmp.begin(), tmp.end(), 0);
      std::copy(tmp.begin(), tmp.end(),
		mwfindex.begin()+wf->GetWFStart(iwf)-start);
    }
    thrust::copy(mwfindex.begin(), mwfindex.end(), t.begin());
    fill(s.begin(), s.end(), 0);
    fill(z.begin(), z.end(), 0.0);
    fill(v.begin(), v.end(), 0.0);
    // baseline subtraction
    transform(y.begin(), y.end(),
	      make_constant_iterator(wf->GetParam("resting_base")),
	      y.begin(), minus<float>());
    if(wfb) copy(y.begin(), y.end(), wfb->wfy.begin()+start);
    int nbase = wf->GetParam("nbase_samples");
    wf->SetWFParam("base",    MeanVals(y, k, z, nbase, 4, wfst));
    wf->SetWFParam("base_rms", RMSVals(y, k, z, nbase, 4, wfst));
    // fit the pz decay constants
    std::vector<float> xtmp(wf->GetParam("nefit_samples"));
    std::iota(xtmp.begin(), xtmp.end(), 0.0);
    std::for_each(xtmp.begin(), xtmp.end(), [&](float& s){ s*=sampling; });
    std::vector<float> xfit((pr.second-pr.first)*xtmp.size());
    std::vector<float> yfit(xfit.size());
    std::vector<int> kfit(xfit.size());
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      int i = xtmp.size() * (iwf-pr.first);
      int j = xtmp.size() + i - 1;
      std::copy(xtmp.begin(), xtmp.end(), xfit.begin()+i);
      std::copy(wf->wfy.begin()+wf->GetWFEnd(iwf)-xtmp.size()-4,
		wf->wfy.begin()+wf->GetWFEnd(iwf)-4, yfit.begin()+i);
      std::fill(kfit.begin()+i, kfit.begin()+j+1, iwf-pr.first);
    }
    device_vector<float> dxfit(xfit.begin(), xfit.end());
    device_vector<float> dyfit(yfit.begin(), yfit.end());
    thrust::copy(kfit.begin(), kfit.end(), s.begin());
    std::vector<float> decay = ExpDecay(dxfit, dyfit, s, z, wfst);
    for(int iwf=pr.first; iwf<pr.second; iwf++)
      wf->SetWFParam(iwf, "decay_const", 1/decay[iwf-pr.first]);
    Trap(y, k, z, s,
         (int) (wf->GetParam("fast_fall") / sampling),
	 (int) (wf->GetParam("fast_flat") / sampling),
	 (int) (wf->GetParam("fast_rise") / sampling));
    thrust::copy(z.begin(), z.end(), wff->wfy.begin()+start);
    // get time points, requires copy of fast trap to cpu above
    exclusive_scan_by_key(k.begin(), k.end(),
			  make_constant_iterator(1), s.begin());
    std::pair<std::vector<float>, std::vector<int> >
      pm = MaxWithIndex(z, k, t, v, s, wf->GetNWaveforms());
    std::vector<float> vb = MeanVals(z, k, v, nbase, 4, wfst);
    std::vector<float> vr =  RMSVals(z, k, v, nbase, 4, wfst);
    float t0_thresh = wf->GetParam("t0_thresh");
    std::vector<int> t0(vb.size());
    std::for_each(vr.begin(), vr.end(), [&](float& s){ s *= t0_thresh; });
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      int jwf = iwf-pr.first;
      int si = wff->GetWFStart(iwf);
      float t1=0.0, t10=0.0, t50=0.0, t90=0.0, t99=0.0;
      for(int i=pm.second[jwf]+wfst[0]-si; i>=si; i--){
	float d = (wff->wfy[i+1] - wff->wfy[i]) / pm.first[jwf];
	if(d == 0.0) d = 1.0e12;
	float w = wff->wfy[i] / pm.first[jwf];
	if(w < 0.01 && t1  == 0.0) t1  = i-si+(w-0.01)/d;
	if(w < 0.1  && t10 == 0.0) t10 = i-si+(w-0.10)/d;
	if(w < 0.5  && t50 == 0.0) t50 = i-si+(w-0.50)/d;
	if(w < 0.9  && t90 == 0.0) t90 = i-si+(w-0.90)/d;
	if(w < 0.99 && t99 == 0.0) t99 = i-si+(w-0.99)/d;
	if(w < vr[jwf]){ t0[jwf] = i-si; break; }
      }
      wf->SetWFParam(iwf, "t0",  (float) t0[jwf]);
      wf->SetWFParam(iwf, "t1",  t1);
      wf->SetWFParam(iwf, "t10", t10);
      wf->SetWFParam(iwf, "t50", t50);
      wf->SetWFParam(iwf, "t90", t90);
      wf->SetWFParam(iwf, "t99", t99);
    }
    // energy estimation with charge trapping correction - David's method 1
    PoleZero(y, k, z, 1-exp(-sampling/wf->GetParam("ct_decay")), false);
    Trap(z, k, v, s,
	 (int) wf->GetParam("slow_nrise"), (int) wf->GetParam("slow_nflat"));
    copy(v.begin(), v.end(), wfc->wfy.begin()+start);
    // charge trapping corrected fixed time pickoff
    int poff = (int)((wf->GetParam("slow_rise") +
		      0.9*wf->GetParam("slow_flat")) / sampling);
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      wf->SetWFParam(iwf, "pickoff", t0[iwf-pr.first] + poff);
      wf->SetWFParam(iwf, "ct1_trappick",
		    wfc->wfy[wfst[iwf-pr.first] + t0[iwf-pr.first] + poff]);
    }
    // vary the charge trapping integration time for optimization
    if(wf->GetParam("nct_steps") > 0.0){
      std::vector<float> tmp(wfc->wfy.size());
      for(int ict=0; ict<(int)wf->GetParam("nct_steps"); ict++){
	PoleZero(y, k, z,
		 1-exp(-sampling/
		       wf->GetParam("ct_decay_"+std::to_string(ict))), false);
	Trap(z, k, v, s,
	     (int)wf->GetParam("slow_nrise"), (int)wf->GetParam("slow_nflat"));
	thrust::copy(v.begin(), v.end(), tmp.begin());
	for(int iwf=pr.first; iwf<pr.second; iwf++)
	  wf->SetWFParam(iwf, "ct1_trappick_"+std::to_string(ict),
			tmp[wfst[iwf-pr.first] + t0[iwf-pr.first] + poff]);
      }
    }
    // standard pole zero correction
    PoleZero(y, k, z, 1-exp(-sampling/wf->GetParam("pz_decay")), true);
    if(wfp) copy(y.begin(), y.end(), wfp->wfy.begin()+start);
    // get the integral for David's 2nd charge trapping method
    int coff = (int) (wf->GetParam("ct_offset") / sampling);
    std::vector<int> ct0=t0, ctn(t0.size(), coff);
    std::vector<float> cti0 = MeanVals(v, k, z, ctn, ct0, wfst);
    std::for_each(ct0.begin(), ct0.end(), [&](int& s){ s+= coff; });
    std::vector<float> cti1 = MeanVals(v, k, z, ctn, ct0, wfst);
    for(int iwf=pr.first; iwf<pr.second; iwf++)
      wf->SetWFParam(iwf, "ct_integral",
		    (cti1[iwf-pr.first]-cti0[iwf-pr.second])*coff);
    // calculate dcr
    int ndcr = (int) wf->GetParam("ndcr_samples");
    std::vector<int> d0(t0.size()), d1(t0.size()), d2(t0.size(), ndcr);
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      d0[iwf-pr.first] = std::max(std::min((int) wf->GetWFParam(iwf, "t99"),
					   wf->GetWFLength(iwf)-2*ndcr-1), 0);
      d1[iwf-pr.first] = wf->GetWFLength(iwf)-ndcr-4;
    }
    std::vector<float> ds0 = MeanVals(y, k, z, d2, d0, wfst, false);
    std::vector<float> ds1 = MeanVals(y, k, z, d2, d1, wfst, false);
    for(int iwf=pr.first; iwf<pr.second; iwf++)
      wf->SetWFParam(iwf, "dcrslope",
		    (ds1[iwf-pr.first]-ds0[iwf-pr.first]) /
		    (wf->GetWFLength(iwf) - d0[iwf-pr.first] - ndcr));
    // trap filter for energy estimation
    Trap(y, k, z, s,
	 (int) wf->GetParam("slow_nrise"), (int) wf->GetParam("slow_nflat"));
    copy(z.begin(), z.end(), wfs->wfy.begin()+start);
    wf->SetWFParam("trap_max", Maxima(z, k, v, s));
    // fixed time energy pickoff
    for(int iwf=pr.first; iwf<pr.second; iwf++){
      double val = wfs->wfy[wfst[iwf-pr.first] + t0[iwf-pr.first] + poff];
      wf->SetWFParam(iwf, "trappick", val);
      val += wf->GetWFParam(iwf, "ct_integral") * wf->GetParam("ct_frac");
      wf->SetWFParam(iwf, "ct2_trappick", val);
    }
    // triangle filter for maximum current estimation
    Trap(y, k, z, s,
	 (int) wf->GetParam("avse_nrise"), (int) wf->GetParam("avse_nflat"));
    if(wfa) copy(z.begin(), z.end(), wfa->wfy.begin()+start);
    wf->SetWFParam("imax", Maxima(z, k, v, s));
  }
  // if copyting wf back to host, populate return map
  if(copy_wf){
    mwf["base_sub"]  = wfb;
    mwf["pz_cor"]    = wfp;
    mwf["pz_ct"]     = wfc;
    mwf["slow_trap"] = wfs;
    mwf["fast_trap"] = wff;
    mwf["avse_trap"] = wfa;
  }
  else{
    delete wfc;
    delete wfs;
    delete wff;
  }
  wf->SetParam("proc_type", 1.0);
  wf->SetParam("proc_time", (float) ptime.RealTime());
  return mwf;
}

double FitExponentials(int np,
		       std::vector<REAL>& x, std::vector<REAL>& y,
		       std::vector<REAL>& p,
		       std::vector<REAL>& param, std::vector<int>& state,
		       std::vector<REAL>& chi2, std::vector<int>& iterations,
		       bool fdecay){
  assert(x.size() == y.size());
  if(x.size() == 0) return 0.0;
  assert(x.size() % np == 0);
  const size_t nfit = x.size() / np;
  assert(p.size() == 2*nfit);
  TStopwatch timer;
  timer.Start();
  param.resize(nfit*2);
  chi2.resize(nfit);
  state.resize(nfit);
  iterations.resize(nfit);
  std::vector<int> fparam(2, 1);
  if(!fdecay) fparam[1] = 0;
  const int status = gpufit(nfit, np, y.data(), 0, EXP_1D, p.data(), 0.001,
			    1000, fparam.data(), LSE, x.size()*sizeof(REAL),
			    reinterpret_cast<char*>(x.data()),
			    param.data(), state.data(),
			    chi2.data(), iterations.data());
  double t = timer.RealTime();
  if(status != ReturnState::OK)
    std::cout << "gpu fitting error, aborting fits" << std::endl;
  std::vector<int> states(5, 0);
  
  std::cout << "Fit " << nfit << " event(s) in "
	    << std::fixed << std::setprecision(2)
	    << t << " seconds: " << nfit/t << " fits/s" << std::endl;
  
  for(std::vector<int>::iterator it=state.begin(); it!=state.end(); it++)
    states[*it] ++;
  
  if(states[0] / (float) nfit > 0.95)
    std::cout << "  " << std::fixed << std::setprecision(2)
	      << 100.*states[0]/nfit << "% of fits converged" << std::endl;
  else{
    std::cout << "  ratio converged     " << (float)states[0]/nfit <<std::endl;
    std::cout << "  ratio max iteration " << (float)states[1]/nfit <<std::endl;
    std::cout << "  ratio singular hess " << (float)states[2]/nfit <<std::endl;
    std::cout << "  ratio neg curvature " << (float)states[3]/nfit <<std::endl;
    std::cout << "  ratio gpu not read  " << (float)states[4]/nfit <<std::endl;
  }
  
  return t;
}

size_t GPUMemory(){
  size_t fmem, tmem;
  hipError_t e = hipMemGetInfo(&fmem, &tmem);
  if(e == hipSuccess) return fmem;
  else return 0;
}
